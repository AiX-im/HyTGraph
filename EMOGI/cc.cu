#include "hip/hip_runtime.h"
/* References:
 *
 *     Hong, Sungpack, et al.
 *     "Accelerating CUDA graph algorithms at maximum warp."
 *     Acm Sigplan Notices 46.8 (2011): 267-276.
 *
 *     Zhen Xu, Xuhao Chen, Jie Shen, Yang Zhang, Cheng Chen, Canqun Yang,
 *     GARDENIA: A Domain-specific Benchmark Suite for Next-generation Accelerators,
 *     ACM Journal on Emerging Technologies in Computing Systems, 2018.
 *
 */

#include "helper_emogi.h"

#define MEM_ALIGN MEM_ALIGN_64

typedef uint64_t EdgeT;

__global__ void kernel_coalesce(bool *curr_visit, bool *next_visit, uint64_t vertex_count, uint64_t *vertexList, EdgeT *edgeList, unsigned long long *comp, bool *changed) {
    const uint64_t tid = blockDim.x * BLOCK_SIZE * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
    const uint64_t warpIdx = tid >> WARP_SHIFT;
    const uint64_t laneIdx = tid & ((1 << WARP_SHIFT) - 1);

    if (warpIdx < vertex_count && curr_visit[warpIdx] == true) {
        const uint64_t start = vertexList[warpIdx];
        const uint64_t shift_start = start & MEM_ALIGN;
        const uint64_t end = vertexList[warpIdx+1];

        for(uint64_t i = shift_start + laneIdx; i < end; i += WARP_SIZE) {
            if (i >= start) {
                unsigned long long comp_src = comp[warpIdx];
                const EdgeT next = edgeList[i];

                unsigned long long comp_next = comp[next];
                unsigned long long comp_target;
                EdgeT next_target;

                if (comp_next != comp_src) {
                    if (comp_src < comp_next) {
                        next_target = next;
                        comp_target = comp_src;
                    }
                    else {
                        next_target = warpIdx;
                        comp_target = comp_next;
                    }

                    atomicMin(&comp[next_target], comp_target);
                    next_visit[next_target] = true;
                    *changed = true;
                }
            }
        }
    }
}

__global__ void kernel_coalesce_chunk(bool *curr_visit, bool *next_visit, uint64_t vertex_count, uint64_t *vertexList, EdgeT *edgeList, unsigned long long *comp, bool *changed) {
    const uint64_t tid = blockDim.x * BLOCK_SIZE * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
    const uint64_t warpIdx = tid >> WARP_SHIFT;
    const uint64_t laneIdx = tid & ((1 << WARP_SHIFT) - 1);
    const uint64_t chunkIdx = warpIdx * CHUNK_SIZE;
    uint64_t chunk_size = CHUNK_SIZE;

    if((chunkIdx + CHUNK_SIZE) > vertex_count) {
        if ( vertex_count > chunkIdx )
            chunk_size = vertex_count - chunkIdx;
        else
            return;
    }

    for(uint32_t i = chunkIdx; i < chunk_size + chunkIdx; i++) {
        if(curr_visit[i]) {
            const uint64_t start = vertexList[i];
            const uint64_t shift_start = start & MEM_ALIGN;
            const uint64_t end = vertexList[i+1];

            for(uint64_t j = shift_start + laneIdx; j < end; j += WARP_SIZE) {
                if (j >= start) {
                    unsigned long long comp_src = comp[i];
                    const EdgeT next = edgeList[j];

                    unsigned long long comp_next = comp[next];
                    unsigned long long comp_target;
                    EdgeT next_target;

                    if (comp_next != comp_src) {
                        if (comp_src < comp_next) {
                            next_target = next;
                            comp_target = comp_src;
                        }
                        else {
                            next_target = i;
                            comp_target = comp_next;
                        }

                        atomicMin(&comp[next_target], comp_target);
                        next_visit[next_target] = true;
                        *changed = true;
                    }
                }
            }
        }
    }
}

int main(int argc, char *argv[]) {
    std::ifstream file;
    std::string vertex_file, edge_file;
    std::string filename;

    bool changed_h, *changed_d;
    bool *curr_visit_d, *next_visit_d, *comp_check;
    int c, arg_num = 0, device = 0;
    impl_type type;
    mem_type mem;
    uint32_t iter, comp_total = 0;
    unsigned long long *comp_d, *comp_h;
    uint64_t *vertexList_h, *vertexList_d;
    EdgeT *edgeList_h, *edgeList_d;
    uint64_t vertex_count, edge_count, vertex_size, edge_size;
    uint64_t typeT;
    uint64_t numblocks, numthreads;

    float milliseconds;

    hipEvent_t start, end;

    while ((c = getopt(argc, argv, "f:t:m:d:h")) != -1) {
        switch (c) {
            case 'f':
                filename = optarg;
                arg_num++;
                break;
            case 't':
                type = (impl_type)atoi(optarg);
                arg_num++;
                break;
            case 'm':
                mem = (mem_type)atoi(optarg);
                arg_num++;
                break;
            case 'd':
                device = atoi(optarg);
                break;
            case 'h':
                printf("8-byte edge CC, only works correctly with undirected graphs\n");
                printf("\t-f | input file name (must end with .bel)\n");
                printf("\t-t | type of CC to run\n");
                printf("\t   | COALESCE = 1, COALESCE_CHUNK = 2\n");
                printf("\t-m | memory allocation\n");
                printf("\t   | GPUMEM = 0, UVM_READONLY = 1, UVM_DIRECT = 2\n");
                printf("\t-h | help message\n");
                return 0;
            case '?':
                break;
            default:
                break;
        }
    }

    if (arg_num < 3) {
        printf("8-byte edge CC, only works correctly with undirected graphs\n");
        printf("\t-f | input file name (must end with .bel)\n");
        printf("\t-t | type of CC to run\n");
        printf("\t   | COALESCE = 1, COALESCE_CHUNK = 2\n");
        printf("\t-m | memory allocation\n");
        printf("\t   | GPUMEM = 0, UVM_READONLY = 1, UVM_DIRECT = 2\n");
        printf("\t-h | help message\n");
        return 0;
    }

    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&end));

    vertex_file = filename + ".col";
    edge_file = filename + ".dst";

    std::cout << filename << std::endl;

    // Read files
    file.open(vertex_file.c_str(), std::ios::in | std::ios::binary);
    if (!file.is_open()) {
        printf("vertex file open failed\n");
        exit(1);
    }

    file.read((char*)(&vertex_count), 8);
    file.read((char*)(&typeT), 8);

    vertex_count--;

    printf("Vertex: %lu, ", vertex_count);
    vertex_size = (vertex_count+1) * sizeof(uint64_t);

    vertexList_h = (uint64_t*)malloc(vertex_size);

    file.read((char*)vertexList_h, vertex_size);
    file.close();

    file.open(edge_file.c_str(), std::ios::in | std::ios::binary);
    if (!file.is_open()) {
        printf("edge file open failed\n");
        exit(1);
    }

    file.read((char*)(&edge_count), 8);
    file.read((char*)(&typeT), 8);

    printf("Edge: %lu\n", edge_count);
    fflush(stdout);
    edge_size = edge_count * sizeof(EdgeT);

    edgeList_h = NULL;

    switch (mem) {
        case GPUMEM:
            edgeList_h = (EdgeT*)malloc(edge_size);
            file.read((char*)edgeList_h, edge_size);
            checkCudaErrors(hipMalloc((void**)&edgeList_d, edge_size));

            break;
        case UVM_READONLY:
            checkCudaErrors(hipMallocManaged((void**)&edgeList_d, edge_size));
            file.read((char*)edgeList_d, edge_size);

            checkCudaErrors(hipMemAdvise(edgeList_d, edge_size, hipMemAdviseSetReadMostly, device));
            break;
        case UVM_DIRECT:
            checkCudaErrors(hipMallocManaged((void**)&edgeList_d, edge_size));
            file.read((char*)edgeList_d, edge_size);

            checkCudaErrors(hipMemAdvise(edgeList_d, edge_size, hipMemAdviseSetAccessedBy, device));
            break;
    }

    file.close();

    // Allocate memory for GPU
    comp_h = (unsigned long long*)malloc(vertex_count * sizeof(unsigned long long));
    comp_check = (bool*)malloc(vertex_count * sizeof(bool));
    checkCudaErrors(hipMalloc((void**)&vertexList_d, vertex_size));
    checkCudaErrors(hipMalloc((void**)&curr_visit_d, vertex_count * sizeof(bool)));
    checkCudaErrors(hipMalloc((void**)&next_visit_d, vertex_count * sizeof(bool)));
    checkCudaErrors(hipMalloc((void**)&comp_d, vertex_count * sizeof(unsigned long long)));
    checkCudaErrors(hipMalloc((void**)&changed_d, sizeof(bool)));

    printf("Allocation finished\n");
    fflush(stdout);

    // Initialize values
    for (uint64_t i = 0; i < vertex_count; i++)
        comp_h[i] = i;

    memset(comp_check, 0, vertex_count * sizeof(bool));

    checkCudaErrors(hipMemset(curr_visit_d, 0x01, vertex_count * sizeof(bool)));
    checkCudaErrors(hipMemset(next_visit_d, 0x00, vertex_count * sizeof(bool)));
    checkCudaErrors(hipMemcpy(comp_d, comp_h, vertex_count * sizeof(uint64_t), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(vertexList_d, vertexList_h, vertex_size, hipMemcpyHostToDevice));

    if (mem == GPUMEM)
        checkCudaErrors(hipMemcpy(edgeList_d, edgeList_h, edge_size, hipMemcpyHostToDevice));

    numthreads = BLOCK_SIZE;

    switch (type) {
        case COALESCE:
            numblocks = ((vertex_count * WARP_SIZE + numthreads) / numthreads);
            break;
        case COALESCE_CHUNK:
            numblocks = ((vertex_count * (WARP_SIZE / CHUNK_SIZE) + numthreads) / numthreads);
            break;
        default:
            fprintf(stderr, "Invalid type\n");
            exit(1);
            break;
    }

    dim3 blockDim(BLOCK_SIZE, (numblocks+BLOCK_SIZE)/BLOCK_SIZE);

    printf("Initialization done\n");
    fflush(stdout);

    iter = 0;

    checkCudaErrors(hipEventRecord(start, 0));

    // Run CC
    do {
        changed_h = false;
        checkCudaErrors(hipMemcpy(changed_d, &changed_h, sizeof(bool), hipMemcpyHostToDevice));

        switch (type) {
            case COALESCE:
                kernel_coalesce<<<blockDim, numthreads>>>(curr_visit_d, next_visit_d, vertex_count, vertexList_d, edgeList_d, comp_d, changed_d);
                break;
            case COALESCE_CHUNK:
                kernel_coalesce_chunk<<<blockDim, numthreads>>>(curr_visit_d, next_visit_d, vertex_count, vertexList_d, edgeList_d, comp_d, changed_d);
                break;
            default:
                fprintf(stderr, "Invalid type\n");
                exit(1);
                break;
        }

        checkCudaErrors(hipMemset(curr_visit_d, 0x00, vertex_count * sizeof(bool)));

        bool *temp = curr_visit_d;
        curr_visit_d = next_visit_d;
        next_visit_d = temp;

        iter++;

        checkCudaErrors(hipMemcpy(&changed_h, changed_d, sizeof(bool), hipMemcpyDeviceToHost));
    } while(changed_h);

    checkCudaErrors(hipEventRecord(end, 0));
    checkCudaErrors(hipEventSynchronize(end));
    checkCudaErrors(hipEventElapsedTime(&milliseconds, start, end));

    checkCudaErrors(hipMemcpy(comp_h, comp_d, vertex_count * sizeof(unsigned long long), hipMemcpyDeviceToHost));

    for (uint64_t i = 0; i < vertex_count; i++) {
        if (comp_check[comp_h[i]] == false) {
            comp_check[comp_h[i]] = true;
            comp_total++;
        }
    }

    printf("total iterations: %u\n", iter);
    printf("total components: %u\n", comp_total);
    printf("total time: %f ms\n", milliseconds);
    fflush(stdout);

    free(vertexList_h);
    if (edgeList_h)
        free(edgeList_h);
    free(comp_check);
    free(comp_h);
    checkCudaErrors(hipFree(vertexList_d));
    checkCudaErrors(hipFree(edgeList_d));
    checkCudaErrors(hipFree(changed_d));
    checkCudaErrors(hipFree(comp_d));
    checkCudaErrors(hipFree(curr_visit_d));
    checkCudaErrors(hipFree(next_visit_d));

    return 0;
}
